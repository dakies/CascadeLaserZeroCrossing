#include "hip/hip_runtime.h"
 // Device code
__global__ void MatAdd(float* C, const float* A, const float* B, int h, int w)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < h * w)
        C[i] = A[i] + B[i];
}

__global__ void MatSub(float* C, const float* A, const float* B, int h, int w)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < h * w)
        C[i] = A[i] - B[i];
}

__global__ void MatTranspose(float* B, const float* A, int h, int w)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < h * w)
        B[(i % w) * h + (i / w)] = A[i]; 
}

//Matrix Multipication 
__global__ void MatMult(float* C,const float* A, const float* B,  int Aw, int Bw)
{
    // Indexes
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int BLOCK_SIZE=16;  
    
    // Shared memory for A and B
    __shared__ float s_A[16][16];
    __shared__ float s_B[16][16];

    int A_start = Aw * BLOCK_SIZE * by;
    int A_stop  = A_start + Aw - 1;
    int A_step  = BLOCK_SIZE;

    int B_start = BLOCK_SIZE * bx;
    int B_step  = BLOCK_SIZE * Bw;

    float s_C = 0;

    for (int a = A_start, b = B_start;a <= A_stop;a += A_step, b += B_step) 
  {
        s_A[ty][tx] = A[a + Aw * ty + tx];
        s_B[tx][ty] = B[b + Bw * tx + ty];

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k)
           s_C += s_A[ty][k] * s_B[k][tx];

        __syncthreads();
    }

    int c = Bw * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + Bw * ty + tx] = s_C;
  
}

//MATCOPY
__global__ void MatCopy(float* B, const float* A, int h, int w)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < h * w)
        B[i] = A[i];
}

__global__ void CalcH(float* B, const float* X){
   // More efficient to calculate simultaneously sincosf(float x, float *sptr, float *cptr)
    sincosf(float X[0], float *B[0], float *B[2])
    X[0] = -X[3] * X[0]
}

__global__ void invS(float* I, float* X){
    I = 1/X
}